/*
 * Copyright (C) Koji Ueno 2012-2013.
 *
 * This file is part of Graph500 Ueno implementation.
 *
 *  Graph500 Ueno implementation is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  Graph500 Ueno implementation is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with Graph500 Ueno implementation.  If not, see <http://www.gnu.org/licenses/>.
 */
#ifndef __STDC_CONSTANT_MACROS
#define __STDC_CONSTANT_MACROS
#endif
#ifndef __STDC_LIMIT_MACROS
#define __STDC_LIMIT_MACROS
#endif
#ifndef __STDC_FORMAT_MACROS
#define __STDC_FORMAT_MACROS
#endif

// C includes
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define _FERMI_ (__CUDA_ARCH__ >= 200)

#define WARP_IDX (blockIdx.x * blockDim.y + threadIdx.y)

#include "parameters.h"

#undef CUDA_CHECK_PRINT_RANK

__device__ MPI_INFO_ON_GPU mpig;

#include "gpu.hpp"
#include "bfs_kernel.hpp"









