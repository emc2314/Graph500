#include "hip/hip_runtime.h"
/*$Id: MarsScan.cu 713 2009-11-08 11:15:14Z wenbinor $*/
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

// includes, kernels
#include "MarsInc.h"

// Define this to more rigorously avoid bank conflicts, 
// even at the lower (root) levels of the tree
// Note that due to the higher addressing overhead, performance 
// is lower with ZERO_BANK_CONFLICTS enabled.  It is provided
// as an example.
//#define ZERO_BANK_CONFLICTS 

// 16 banks on G80
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS + (index) >> (2*LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS)
#endif

///////////////////////////////////////////////////////////////////////////////
// Work-efficient compute implementation of scan, one thread per 2 elements
// Work-efficient: O(log(n)) steps, and O(n) adds.
// Also shared storage efficient: Uses n + n/NUM_BANKS shared memory -- no ping-ponging
// Also avoids most bank conflicts using single-element offsets every NUM_BANKS elements.
//
// In addition, If ZERO_BANK_CONFLICTS is defined, uses 
//     n + n/NUM_BANKS + n/(NUM_BANKS*NUM_BANKS) 
// shared memory. If ZERO_BANK_CONFLICTS is defined, avoids ALL bank conflicts using 
// single-element offsets every NUM_BANKS elements, plus additional single-element offsets 
// after every NUM_BANKS^2 elements.
//
// Uses a balanced tree type algorithm.  See Blelloch, 1990 "Prefix Sums 
// and Their Applications", or Prins and Chatterjee PRAM course notes:
// http://www.cs.unc.edu/~prins/Classes/203/Handouts/pram.pdf
// 
// This work-efficient version is based on the algorithm presented in Guy Blelloch's
// excellent paper "Prefix sums and their applications".
// http://www-2.cs.cmu.edu/afs/cs.cmu.edu/project/scandal/public/papers/CMU-CS-90-190.html
//
// Pro: Work Efficient, very few bank conflicts (or zero if ZERO_BANK_CONFLICTS is defined)
// Con: More instructions to compute bank-conflict-free shared memory addressing,
// and slightly more shared memory storage used.
//

template <bool isNP2>
__device__ void loadSharedChunkFromMem(int *s_data,
                                       const int *g_idata, 
                                       int n, int baseIndex,
                                       int& ai, int& bi, 
                                       int& mem_ai, int& mem_bi, 
                                       int& bankOffsetA, int& bankOffsetB)
{
    int thid = threadIdx.x;
    mem_ai = baseIndex + threadIdx.x;
    mem_bi = mem_ai + blockDim.x;

    ai = thid;
    bi = thid + blockDim.x;

    // compute spacing to avoid bank conflicts
    bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    bankOffsetB = CONFLICT_FREE_OFFSET(bi);

    // Cache the computational window in shared memory
    // pad values beyond n with zeros
    s_data[ai + bankOffsetA] = g_idata[mem_ai]; 
    
    if (isNP2) // compile-time decision
    {
        s_data[bi + bankOffsetB] = (bi < n) ? g_idata[mem_bi] : 0; 
    }
    else
    {
        s_data[bi + bankOffsetB] = g_idata[mem_bi]; 
    }
}

template <bool isNP2>
__device__ void storeSharedChunkToMem(int* g_odata, 
                                      const int* s_data,
                                      int n, 
                                      int ai, int bi, 
                                      int mem_ai, int mem_bi,
                                      int bankOffsetA, int bankOffsetB)
{
    __syncthreads();

    // write results to global memory
    g_odata[mem_ai] = s_data[ai + bankOffsetA]; 
    if (isNP2) // compile-time decision
    {
        if (bi < n)
            g_odata[mem_bi] = s_data[bi + bankOffsetB]; 
    }
    else
    {
        g_odata[mem_bi] = s_data[bi + bankOffsetB]; 
    }
}

template <bool storeSum>
__device__ void clearLastElement(int* s_data, 
                                 int *g_blockSums, 
                                 int blockIndex)
{
    if (threadIdx.x == 0)
    {
        int index = (blockDim.x << 1) - 1;
        index += CONFLICT_FREE_OFFSET(index);
        
        if (storeSum) // compile-time decision
        {
            // write this block's total sum to the corresponding index in the blockSums array
            g_blockSums[blockIndex] = s_data[index];
        }

        // zero the last element in the scan so it will propagate back to the front
        s_data[index] = 0;
    }
}



__device__ unsigned int buildSum(int *s_data)
{
    unsigned int thid = threadIdx.x;
    unsigned int stride = 1;
    
    // build the sum in place up the tree
    for (int d = blockDim.x; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)      
        {
            int i  = __mul24(__mul24(2, stride), thid);
            int ai = i + stride - 1;
            int bi = ai + stride;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            s_data[bi] += s_data[ai];
        }

        stride *= 2;
    }

    return stride;
}

__device__ void scanRootToLeaves(int *s_data, unsigned int stride)
{
     unsigned int thid = threadIdx.x;

    // traverse down the tree building the scan in place
    for (int d = 1; d <= blockDim.x; d *= 2)
    {
        stride >>= 1;

        __syncthreads();

        if (thid < d)
        {
            int i  = __mul24(__mul24(2, stride), thid);
            int ai = i + stride - 1;
            int bi = ai + stride;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            int t  = s_data[ai];
            s_data[ai] = s_data[bi];
            s_data[bi] += t;
        }
    }
}

template <bool storeSum>
__device__ void prescanBlock(int *data, int blockIndex, int *blockSums)
{
    int stride = buildSum(data);               // build the sum in place up the tree
    clearLastElement<storeSum>(data, blockSums, 
                               (blockIndex == 0) ? blockIdx.x : blockIndex);
    scanRootToLeaves(data, stride);            // traverse down tree to build the scan 
}

template <bool storeSum, bool isNP2>
__global__ void prescan(int *g_odata, 
                        const int *g_idata, 
                        int *g_blockSums, 
                        int n, 
                        int blockIndex, 
                        int baseIndex)
{
    int ai, bi, mem_ai, mem_bi, bankOffsetA, bankOffsetB;
    extern __shared__ int s_data[];

    // load data into shared memory
    loadSharedChunkFromMem<isNP2>(s_data, g_idata, n, 
                                  (baseIndex == 0) ? 
                                  __mul24(blockIdx.x, (blockDim.x << 1)):baseIndex,
                                  ai, bi, mem_ai, mem_bi, 
                                  bankOffsetA, bankOffsetB); 
    // scan the data in each block
    prescanBlock<storeSum>(s_data, blockIndex, g_blockSums); 
    // write results to device memory
    storeSharedChunkToMem<isNP2>(g_odata, s_data, n, 
                                 ai, bi, mem_ai, mem_bi, 
                                 bankOffsetA, bankOffsetB);  
}


__global__ void uniformAdd(int *g_data, 
                           int *uniforms, 
                           int n, 
                           int blockOffset, 
                           int baseIndex)
{
    __shared__ int uni;
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];
    
    unsigned int address = __mul24(blockIdx.x, (blockDim.x << 1)) + baseIndex + threadIdx.x; 

    __syncthreads();
    
    // note two adds per thread
    g_data[address]              += uni;
    g_data[address + blockDim.x] += (threadIdx.x + blockDim.x < n) * uni;
}


inline bool 
isPowerOfTwo(int n)
{
    return ((n&(n-1))==0) ;
}

inline int 
floorPow2(int n)
{
#ifdef WIN32
    // method 2
    return 1 << (int)logb((float)n);
#else
    // method 1
    // int nf = (int)n;
    // return 1 << (((*(int*)&nf) >> 23) - 127); 
    int exp;
    frexp((double)n, &exp);
    return 1 << (exp - 1);
#endif
}

#define BLOCK_SIZE 256

int** g_scanBlockSums;
unsigned int g_numEltsAllocated = 0;
unsigned int g_numLevelsAllocated = 0;



void preallocBlockSums(unsigned int maxNumElements)
{
//    assert(g_numEltsAllocated == 0); // shouldn't be called 

    g_numEltsAllocated = maxNumElements;

    unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
    unsigned int numElts = maxNumElements;

    int level = 0;

    do
    {       
        unsigned int numBlocks = 
            max(1, (int)ceil((int)numElts / (2.f * blockSize)));
        if (numBlocks > 1)
        {
            level++;
        }
        numElts = numBlocks;
    } while (numElts > 1);

    g_scanBlockSums = (int**) malloc(level * sizeof(int*));
    g_numLevelsAllocated = level;
    
    numElts = maxNumElements;
    level = 0;
    
    do
    {       
        unsigned int numBlocks = 
            max(1, (int)ceil((int)numElts / (2.f * blockSize)));
        if (numBlocks > 1) 
        {
            CUDA_SAFE_CALL(hipMalloc((void**) &g_scanBlockSums[level++],  
                                      numBlocks * sizeof(int)));
        }
        numElts = numBlocks;
    } while (numElts > 1);

    CUT_CHECK_ERROR("preallocBlockSums");
}

void deallocBlockSums()
{
    for (int i = 0; i < g_numLevelsAllocated; i++)
    {
        hipFree(g_scanBlockSums[i]);
    }

    CUT_CHECK_ERROR("deallocBlockSums");
    
    free((void**)g_scanBlockSums);

    g_scanBlockSums = 0;
    g_numEltsAllocated = 0;
    g_numLevelsAllocated = 0;
}

void saven_initialPrefixSum(unsigned int maxNumElements)
{
	if(g_numEltsAllocated == 0)
		preallocBlockSums(maxNumElements);
	else
		if(g_numEltsAllocated>maxNumElements)
		{
			deallocBlockSums();
			preallocBlockSums(maxNumElements);
		}
	
}

void prescanArrayRecursive(int *outArray, 
                           const int *inArray, 
                           int numElements, 
                           int level)
{
    unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
    unsigned int numBlocks = 
        max(1, (int)ceil((int)numElements / (2.f * blockSize)));
    unsigned int numThreads;

    if (numBlocks > 1)
        numThreads = blockSize;
    else if (isPowerOfTwo(numElements))
        numThreads = numElements / 2;
    else
        numThreads = floorPow2(numElements);

    unsigned int numEltsPerBlock = numThreads * 2;

    // if this is a non-power-of-2 array, the last block will be non-full
    // compute the smallest power of 2 able to compute its scan.
    unsigned int numEltsLastBlock = 
        numElements - (numBlocks-1) * numEltsPerBlock;
    unsigned int numThreadsLastBlock = max(1, numEltsLastBlock / 2);
    unsigned int np2LastBlock = 0;
    unsigned int sharedMemLastBlock = 0;
    
    if (numEltsLastBlock != numEltsPerBlock)
    {
        np2LastBlock = 1;

        if(!isPowerOfTwo(numEltsLastBlock))
            numThreadsLastBlock = floorPow2(numEltsLastBlock);    
        
        unsigned int extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
        sharedMemLastBlock = 
            sizeof(int) * (2 * numThreadsLastBlock + extraSpace);
    }

    // padding space is used to avoid shared memory bank conflicts
    unsigned int extraSpace = numEltsPerBlock / NUM_BANKS;
    unsigned int sharedMemSize = 
        sizeof(int) * (numEltsPerBlock + extraSpace);

#ifdef DEBUG
    if (numBlocks > 1)
    {
        assert(g_numEltsAllocated >= numElements);
    }
#endif

    // setup execution parameters
    // if NP2, we process the last block separately
    dim3  grid(max(1, numBlocks - np2LastBlock), 1, 1); 
    dim3  threads(numThreads, 1, 1);

    // make sure there are no CUDA errors before we start
    CUT_CHECK_ERROR("prescanArrayRecursive before kernels");

    // execute the scan
    if (numBlocks > 1)
    {
        prescan<true, false><<< grid, threads, sharedMemSize >>>(outArray, 
                                                                 inArray, 
                                                                 g_scanBlockSums[level],
                                                                 numThreads * 2, 0, 0);
        CUT_CHECK_ERROR("prescanWithBlockSums");
        if (np2LastBlock)
        {
            prescan<true, true><<< 1, numThreadsLastBlock, sharedMemLastBlock >>>
                (outArray, inArray, g_scanBlockSums[level], numEltsLastBlock, 
                 numBlocks - 1, numElements - numEltsLastBlock);
            CUT_CHECK_ERROR("prescanNP2WithBlockSums");
        }

        // After scanning all the sub-blocks, we are mostly done.  But now we 
        // need to take all of the last values of the sub-blocks and scan those.  
        // This will give us a new value that must be sdded to each block to 
        // get the final results.
        // recursive (CPU) call
        prescanArrayRecursive(g_scanBlockSums[level], 
                              g_scanBlockSums[level], 
                              numBlocks, 
                              level+1);

        uniformAdd<<< grid, threads >>>(outArray, 
                                        g_scanBlockSums[level], 
                                        numElements - numEltsLastBlock, 
                                        0, 0);
        CUT_CHECK_ERROR("uniformAdd");
        if (np2LastBlock)
        {
            uniformAdd<<< 1, numThreadsLastBlock >>>(outArray, 
                                                     g_scanBlockSums[level], 
                                                     numEltsLastBlock, 
                                                     numBlocks - 1, 
                                                     numElements - numEltsLastBlock);
            CUT_CHECK_ERROR("uniformAdd");
        }
    }
    else if (isPowerOfTwo(numElements))
    {
        prescan<false, false><<< grid, threads, sharedMemSize >>>(outArray, inArray,
                                                                  0, numThreads * 2, 0, 0);
        CUT_CHECK_ERROR("prescan");
    }
    else
    {
         prescan<false, true><<< grid, threads, sharedMemSize >>>(outArray, inArray, 
                                                                  0, numElements, 0, 0);
         CUT_CHECK_ERROR("prescanNP2");
    }
}

void prescanArray(int *outArray, int *inArray, int numElements)
{
    prescanArrayRecursive(outArray, inArray, numElements, 0);
}

int prefexSum( int* d_inArr, int* d_outArr, int numRecords )
{	
	preallocBlockSums(numRecords);
	prescanArray( d_outArr, d_inArr, numRecords );
	deallocBlockSums();	

	int* h_outLast = ( int* )malloc( sizeof( int ) );
	CUDA_SAFE_CALL( hipMemcpy( h_outLast, d_outArr+numRecords-1, sizeof(int),
                                hipMemcpyDeviceToHost) );
	int* h_inLast = ( int* )malloc( sizeof( int ) );
	CUDA_SAFE_CALL( hipMemcpy( h_inLast, d_inArr+numRecords-1, sizeof(int),
                                hipMemcpyDeviceToHost) );

	unsigned int sum = *h_outLast + *h_inLast;

	free( h_outLast );
	free( h_inLast );
	
	return sum;
}
#endif // _PRESCAN_CU_
