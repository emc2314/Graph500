/***********************************************************************
 	graphgpu
	Authors: Koichi Shirahata, Hitoshi Sato, Toyotaro Suzumura, and Satoshi Matsuoka

This software is licensed under Apache License, Version 2.0 (the  "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
***********************************************************************/

#ifndef __REDUCE1_CU__
#define __REDUCE1_CU__

#include "MarsInc.h"
#include "global.h"

//#define _DEBUG_REDUCE

__device__ void REDUCE_COUNT_FUNC1//(void* key, void* vals, size_t keySize, size_t valCount)
{
  for(int i = 0; i < valCount; i++) {
    EMIT_COUNT_FUNC(sizeof(int), sizeof(RWR_VAL_T));
  }
}

__device__ void REDUCE_FUNC1//(void* key, void* vals, size_t keySize, size_t valCount)
{
  int i;
  float cur_rank = 0;
  
  int *dst_nodes_list = NULL;
  size_t valSize = valCount * sizeof(int); 

  dst_nodes_list = (int*)malloc(valSize);
  int dst_nodes_list_size = 0;
  
  for(i = 0; i < valCount; i++) {
    RWR_VAL_T* iVal = (RWR_VAL_T*)GET_VAL_FUNC(vals, i);
    if(iVal->is_v == true) { // vector : VALUE
      cur_rank = iVal->dst;
    }
    else {  // edge ROWID
      dst_nodes_list[dst_nodes_list_size] = (int)iVal->dst;
      dst_nodes_list_size++;
    }
  }
  
  // add random coeff
  RWR_VAL_T* o_val = (RWR_VAL_T*)GET_OUTPUT_BUF(0);
  o_val->is_v = false;
  o_val->dst = cur_rank;
  EMIT_FUNC(key, o_val, sizeof(RWR_KEY_T), sizeof(RWR_VAL_T));

  int outdeg = dst_nodes_list_size;
  if(outdeg > 0) {
    cur_rank = cur_rank / (float)outdeg;
  }
  for(i = 0; i < outdeg; i++) {
    RWR_VAL_T* o_val = (RWR_VAL_T*)GET_OUTPUT_BUF(0);
    o_val->is_v = true;
    o_val->dst = cur_rank;	  
    EMIT_FUNC(&dst_nodes_list[i], o_val, sizeof(RWR_KEY_T), sizeof(RWR_VAL_T));
  }

  free(dst_nodes_list);

}

#endif //__REDUCE1_CU__
