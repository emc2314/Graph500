#include "hip/hip_runtime.h"
/*$Id: main.cu 738 2009-11-13 16:08:10Z wenbinor $*/
/**
 *This is the source code for Mars, a MapReduce framework on graphics
 *processors.
 *Developers: Wenbin Fang (HKUST), Bingsheng He (Microsoft Research Asia)
 *Naga K. Govindaraju (Microsoft Corp.), Qiong Luo (HKUST), Tuyong Wang (Sina.com).
 *If you have any question on the code, please contact us at 
 *           wenbin@cse.ust.hk or savenhe@microsoft.com
 *
 *The license is a free non-exclusive, non-transferable license to reproduce, 
 *use, modify and display the source code version of the Software, with or 
 *without modifications solely for non-commercial research, educational or 
 *evaluation purposes. The license does not entitle Licensee to technical support, 
 *telephone assistance, enhancements or updates to the Software. All rights, title 
 *to and ownership interest in Mars, including all intellectual property rights 
 *therein shall remain in HKUST.
 */

/*********************************************************************************
 *Similarity Score (SS): It is used in web document clustering. 
 *The characteristics of a document are represented as a feature vector. 
 *Given two document features, a and b , the similarity score between these two
 *documents is defined to be a.b/(|a|.|b|).
 *This application computes the pair-wise similarity score for a set of documents. 
 *Each Map computes the similarity score for two documents. 
 *It outputs the intermediate pair of the score as the key and the pair of the two 
 *document IDs as the value. No Reduce stage is required.
 *********************************************************************************/

#include "MarsInc.h"
#include "global.h"

//#define __OUTPUT__

static float *GenMatrix(int M_ROW_COUNT, int M_COL_COUNT)
{
	float *matrix = (float*)malloc(sizeof(float)*M_ROW_COUNT*M_COL_COUNT);

	srand(time(0));
	for (int i = 0; i < M_ROW_COUNT; i++)
		for (int j = 0; j < M_COL_COUNT; j++)
			matrix[i*M_COL_COUNT+j] = (float)(rand() % 100);

	return matrix;
}


#define SMALL_NUM	0.0000001
int cmp(const void* a, const void* b)
{
	float aa = ((SS_VAL_T*)a)->result;
	float bb = ((SS_VAL_T*)b)->result;
	if (abs(aa-bb) < SMALL_NUM) return 0;
	if (aa < bb) return 1;
	return -1;
}


void printFun(void* key, void* val, int keySize, int valSize)
{
	float* result = (float*)key;
	int2* pos = (int2*)val;

	printf("GPU:%f - (%d, %d)\n", *result, pos->x, pos->y);
}

void validate(float* matrix, Spec_t* spec, int row_num, int col_num)
{
	SS_VAL_T* result = (SS_VAL_T*)malloc(row_num*row_num*sizeof(SS_VAL_T));
	int count = 0;
	for (int i = 0; i < row_num; i++)
	{
		int doc1 = i;
		for (int j = i+1; j < row_num; j++)
		{
             	     float up = 0;
      	             float downa = 0;
   		     float downb = 0;

			int doc2 = j;
        		for (int k= 0; k < col_num; k++){
         			up +=       matrix[doc1 *col_num + k] * matrix[doc2 *col_num + k];
         			downa += matrix[doc1 *col_num + k] * matrix[doc1 *col_num + k];
         			downb += matrix[doc2 *col_num + k] * matrix[doc2 *col_num + k];
        		}
			result[count].doc1 = doc1;
			result[count].doc2 = doc2;
       			result[count++].result = up/(sqrtf(downa)*sqrtf(downb));
		}
	}

	qsort(result, count, sizeof(SS_VAL_T), cmp);

	int displayNum  = 10;
	
	for (int i =0; i < displayNum; i++)
		printf("CPU:%f- (%d, %d)\n", result[i].result, result[i].doc1, result[i].doc2);

	printf("-------------------\n");
	PrintOutputRecords(spec, displayNum, printFun);
	free(result);
}

//--------------------------------------------------------------------
//usage: SimilarityScore rowNum colNum 
//param: rowNum
//param: colNum
//--------------------------------------------------------------------
int main( int argc, char** argv) 
{
	if (argc != 3)
	{
		printf("usage: %s rowNum colNum\n", argv[0]);
		exit(0);
	}

	Spec_t *spec = GetDefaultSpec();
#ifdef __OUTPUT__
	spec->outputToHost = 1;
#endif
	spec->workflow = MAP_GROUP;

	int M_ROW_COUNT = atoi(argv[1]);
	int M_COL_COUNT = atoi(argv[2]);
	
	//----------------------------------------------------------
	//load matrix
	//----------------------------------------------------------
	float *matrix = GenMatrix(M_ROW_COUNT, M_COL_COUNT);

	DoLog("load matrice...");

	int matrixSize = sizeof(float)*M_ROW_COUNT*M_COL_COUNT;

	float *d_matrix = NULL;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_matrix, matrixSize));
	CUDA_SAFE_CALL(hipMemcpy(d_matrix, matrix, matrixSize, hipMemcpyHostToDevice));

	//-----------------------------------------------------------
	//make map input
	//-----------------------------------------------------------
	SS_KEY_T ptr;
	ptr.matrix = d_matrix;
	SS_VAL_T doc_info;
	doc_info.dim = M_COL_COUNT;
	doc_info.result = 0.0f;
	TimeVal_t alltimer;
	startTimer(&alltimer);
	for (int i = 0; i < M_ROW_COUNT; i++)
	{
		doc_info.doc1 = i;

		for (int j = i+1; j < M_ROW_COUNT; j++)
		{
			doc_info.doc2 = j;
			AddMapInputRecord(spec, &ptr, &doc_info, 
				sizeof(SS_KEY_T), sizeof(SS_VAL_T));	
		}
	}

	//------------------------------------------------------------
	//main MapReduce procedure
	//------------------------------------------------------------
	MapReduce(spec);

	endTimer("all-test", &alltimer);

	//------------------------------------------------------------
	//further process
	//**Please turn on spec->outputToHost
	//------------------------------------------------------------
#ifdef __OUTPUT__
	validate(matrix, spec, M_ROW_COUNT, M_COL_COUNT);
#endif

	//------------------------------------------------------------
	//finish
	//------------------------------------------------------------
	FinishMapReduce(spec);

	return 0;
}
