/*$Id: reduce.cu 731 2009-11-13 14:45:27Z wenbinor $*/
/**
 *This is the source code for Mars, a MapReduce framework on graphics
 *processors.
 *Developers: Wenbin Fang (HKUST), Bingsheng He (Microsoft Research Asia)
 *Naga K. Govindaraju (Microsoft Corp.), Qiong Luo (HKUST), Tuyong Wang (Sina.com).
 *If you have any question on the code, please contact us at 
 *           wenbin@cse.ust.hk or savenhe@microsoft.com
 *
 *The license is a free non-exclusive, non-transferable license to reproduce, 
 *use, modify and display the source code version of the Software, with or 
 *without modifications solely for non-commercial research, educational or 
 *evaluation purposes. The license does not entitle Licensee to technical support, 
 *telephone assistance, enhancements or updates to the Software. All rights, title 
 *to and ownership interest in Mars, including all intellectual property rights 
 *therein shall remain in HKUST.
 */

#ifndef __REDUCE_CU__
#define __REDUCE_CU__

#include "MarsInc.h"
#include "global.h"

__device__ void REDUCE_COUNT_FUNC//(void* key, void* vals, size_t keySize, size_t valCount)
{
	EMIT_COUNT_FUNC(sizeof(PVC_KEY_T), sizeof(PVC_VAL_T));
}

__device__ void REDUCE_FUNC//(void* key, void* vals, size_t keySize, size_t valCount)
{
	PVC_VAL_T* pVal = (PVC_VAL_T*)vals;
	pVal->phase = 1;
	EMIT_FUNC(key, vals, sizeof(PVC_KEY_T), sizeof(PVC_VAL_T));
}
#endif //__REDUCE_CU__
