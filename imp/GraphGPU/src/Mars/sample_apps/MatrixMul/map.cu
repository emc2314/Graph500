#include "hip/hip_runtime.h"
/*$Id: map.cu 720 2009-11-10 10:13:52Z wenbinor $*/
/**
 *This is the source code for Mars, a MapReduce framework on graphics
 *processors.
 *Developers: Wenbin Fang (HKUST), Bingsheng He (Microsoft Research Asia)
 *Naga K. Govindaraju (Microsoft Corp.), Qiong Luo (HKUST), Tuyong Wang (Sina.com).
 *If you have any question on the code, please contact us at 
 *           wenbin@cse.ust.hk or savenhe@microsoft.com
 *
 *The license is a free non-exclusive, non-transferable license to reproduce, 
 *use, modify and display the source code version of the Software, with or 
 *without modifications solely for non-commercial research, educational or 
 *evaluation purposes. The license does not entitle Licensee to technical support, 
 *telephone assistance, enhancements or updates to the Software. All rights, title 
 *to and ownership interest in Mars, including all intellectual property rights 
 *therein shall remain in HKUST.
 */

#ifndef __MAP_CU__
#define __MAP_CU__

#include "MarsInc.h"
#include "global.h"

__device__ float operator*(float4 a, float4 b)
{
	return (a.x*b.x+a.y*b.y+a.z*b.z+a.w*b.w);
}

__device__ void MAP_COUNT_FUNC//(void *key, void *val, size_t keySize, size_t valSize)
{			
	EMIT_INTER_COUNT_FUNC(sizeof(float), sizeof(int2));
}

__device__ void MAP_FUNC//(void *key, void val, size_t keySize, size_t valSize)
{
	MM_KEY_T* pKey = ((MM_KEY_T*)key);
	MM_VAL_T* pVal = ((MM_VAL_T*)val);

	int rowId = pVal->row;
	int colId = pVal->col;

	int M_COL_COUNT = pVal->col_dim;

	float4 *matrix1 = (float4*)(pKey->matrix1+rowId*M_COL_COUNT);
	float4 *matrix2 = (float4*)(pKey->matrix2+colId*M_COL_COUNT);

	float newVal = 0.0f;

	int col4 = M_COL_COUNT >> 2;
	int remainder = M_COL_COUNT & 0x00000003;

	for (int i = 0; i < col4; i++)
	{
		float4 v1 = matrix1[i];
		float4 v2 = matrix2[i];

		newVal += v1.x * v2.x;
		newVal += v1.y * v2.y;
		newVal += v1.z * v2.z;
		newVal += v1.w * v2.w;
	}

	float *rMatrix1 = (float*)(matrix1+col4);
	float *rMatrix2 = (float*)(matrix2+col4);

	for (int i = 0; i < remainder; i++)
	{
		float f1 = rMatrix1[i];
		float f2 = rMatrix2[i];
		newVal += (f1 * f2);
	}

	float* o_result = (float*)GET_OUTPUT_BUF(0);
	*o_result = newVal;
	int2* o_pos = (int2*)GET_OUTPUT_BUF(sizeof(float));
	o_pos->x = rowId;
	o_pos->y = colId;
	EMIT_INTERMEDIATE_FUNC(o_result, o_pos, sizeof(float), sizeof(int2));			 
}

#endif //__MAP_CU__
