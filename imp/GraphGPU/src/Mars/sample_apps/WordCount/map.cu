/*$Id: map.cu 755 2009-11-18 13:22:54Z wenbinor $*/
/**
 *This is the source code for Mars, a MapReduce framework on graphics
 *processors.
 *Developers: Wenbin Fang (HKUST), Bingsheng He (Microsoft Research Asia)
 *Naga K. Govindaraju (Microsoft Corp.), Qiong Luo (HKUST), Tuyong Wang (Sina.com).
 *If you have any question on the code, please contact us at 
 *           wenbin@cse.ust.hk or savenhe@microsoft.com
 *
 *The license is a free non-exclusive, non-transferable license to reproduce, 
 *use, modify and display the source code version of the Software, with or 
 *without modifications solely for non-commercial research, educational or 
 *evaluation purposes. The license does not entitle Licensee to technical support, 
 *telephone assistance, enhancements or updates to the Software. All rights, title 
 *to and ownership interest in Mars, including all intellectual property rights 
 *therein shall remain in HKUST.
 */

#ifndef __MAP_CU__
#define __MAP_CU__

#include "MarsInc.h"
#include "global.h"

__device__ int hash_func(char* str, int len)
{
	int hash, i;
	for (i = 0, hash=len; i < len; i++)
		hash = (hash<<4)^(hash>>28)^str[i];
	return hash;
}

__device__ void MAP_COUNT_FUNC//(void *key, void *val, size_t keySize, size_t valSize)
{
	WC_KEY_T* pKey = (WC_KEY_T*)key;
	WC_VAL_T* pVal = (WC_VAL_T*)val;

	char* ptrBuf = pKey->file + pVal->line_offset;
	int line_size = pVal->line_size;

	char* p = ptrBuf;
	int lsize = 0;
	int wsize = 0;
	char* start = ptrBuf;

	while(1)
	{
		for (; *p >= 'A' && *p <= 'Z'; p++, lsize++);
		*p = '\0';
		++p;
		++lsize;
		wsize = (int)(p - start);
		if (wsize > 6)
		{
			//printf("%s, wsize:%d\n", start, wsize);	
			EMIT_INTER_COUNT_FUNC(wsize, sizeof(int));
		}
		for (; (lsize < line_size) && (*p < 'A' || *p > 'Z'); p++, lsize++);
		if (lsize >= line_size) break;
		start = p;
	}
}

__device__ void MAP_FUNC//(void *key, void val, size_t keySize, size_t valSize)
{
	WC_KEY_T* pKey = (WC_KEY_T*)key;
	WC_VAL_T* pVal = (WC_VAL_T*)val;

	char* filebuf = pKey->file;
	char* ptrBuf = filebuf + pVal->line_offset;
	int line_size = pVal->line_size;

	char* p = ptrBuf;
	char* start = ptrBuf;
	int lsize = 0;
	int wsize = 0;

	while(1)
	{
		for (; *p >= 'A' && *p <= 'Z'; p++, lsize++);
		*p = '\0';
		++p;
		++lsize;
		wsize = (int)(p - start);
		int* o_val = (int*)GET_OUTPUT_BUF(0);
		*o_val = wsize;
		if (wsize > 6) 
		{
			//printf("%s, %d\n", start, wsize);	
			EMIT_INTERMEDIATE_FUNC(start, o_val, wsize, sizeof(int));
		}
		for (; (lsize < line_size) && (*p < 'A' || *p > 'Z'); p++, lsize++);
		if (lsize >= line_size) break;
		start = p;	
	}
}
#endif //__MAP_CU__
