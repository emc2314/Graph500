#include "hip/hip_runtime.h"
/*$Id: main.cu 755 2009-11-18 13:22:54Z wenbinor $*/
/**
 *This is the source code for Mars, a MapReduce framework on graphics
 *processors.
 *Developers: Wenbin Fang (HKUST), Bingsheng He (Microsoft Research Asia)
 *Naga K. Govindaraju (Microsoft Corp.), Qiong Luo (HKUST), Tuyong Wang (Sina.com).
 *If you have any question on the code, please contact us at 
 *           wenbin@cse.ust.hk or savenhe@microsoft.com
 *
 *The license is a free non-exclusive, non-transferable license to reproduce, 
 *use, modify and display the source code version of the Software, with or 
 *without modifications solely for non-commercial research, educational or 
 *evaluation purposes. The license does not entitle Licensee to technical support, 
 *telephone assistance, enhancements or updates to the Software. All rights, title 
 *to and ownership interest in Mars, including all intellectual property rights 
 *therein shall remain in HKUST.
 */

/******************************************************************
 *WordCount (WC): It counts the number of occurrences for each word in a file. Each Map
 * task processes a portion of the input file and emits intermediate data pairs, each of which consists
 * of a word as the key and a value of 1 for the occurrence. Group is required, and no reduce is
 * needed, because the Mars runtime provides the size of each group, after the Group stage.
 ******************************************************************/

#include "MarsInc.h"
#include "global.h"
#include <ctype.h>

#define __OUTPUT__

void validate(char* h_filebuf, Spec_t* spec, int num)
{
	char* key = (char*)spec->outputKeys;
	char* val = (char*)spec->outputVals;
	int4* offsetSizes = (int4*)spec->outputOffsetSizes;
	int2* range = (int2*)spec->outputKeyListRange;

	printf("# of words:%d\n", spec->outputDiffKeyCount);
	if (num > spec->outputDiffKeyCount) num = spec->outputDiffKeyCount;
	for (int i = 0; i < num; i++)
	{
		int keyOffset = offsetSizes[range[i].x].x;
		int valOffset = offsetSizes[range[i].x].z;
		char* word = key + keyOffset;
		int wordsize = *(int*)(val + valOffset);
		printf("%s - size: %d - count: %d\n", word, wordsize, range[i].y - range[i].x);
	}
}

//-----------------------------------------------------------------------
//usage: WordCount datafile
//param: datafile 
//-----------------------------------------------------------------------
int main( int argc, char** argv) 
{
	if (argc != 2)
	{
		printf("usage: %s datafile\n", argv[0]);
		exit(-1);	
	}
	
	Spec_t *spec = GetDefaultSpec();
	spec->workflow = MAP_GROUP;
#ifdef __OUTPUT__
	spec->outputToHost = 1;
#endif

	TimeVal_t allTimer;
	startTimer(&allTimer);

	TimeVal_t preTimer;
	startTimer(&preTimer);

	FILE* fp = fopen(argv[1], "r");
	fseek(fp, 0, SEEK_END);
	int fileSize = ftell(fp) + 1;
	rewind(fp);
	char* h_filebuf = (char*)malloc(fileSize);
	char* d_filebuf = NULL;
	fread(h_filebuf, fileSize, 1, fp);
	CUDA_SAFE_CALL(hipMalloc((void**)&d_filebuf, fileSize));	
	fclose(fp);

	WC_KEY_T key;
	key.file = d_filebuf;

	for (int i = 0; i < fileSize; i++)
		h_filebuf[i] = toupper(h_filebuf[i]);

	WC_VAL_T val;
	int offset = 0;
	char* p = h_filebuf;
	char* start = h_filebuf;
	while (1)
	{
		int blockSize = 2048;
		if (offset + blockSize > fileSize) blockSize = fileSize - offset;
		p += blockSize;
		for (; *p >= 'A' && *p <= 'Z'; p++);
			
		if (*p != '\0') 
		{
			*p = '\0'; 
			++p;
			blockSize = (int)(p - start);
			val.line_offset = offset;
			val.line_size = blockSize;
			AddMapInputRecord(spec, &key, &val, sizeof(WC_KEY_T), sizeof(WC_VAL_T));	
			offset += blockSize;
			start = p;
		}
		else
		{
			*p = '\0'; 
			blockSize = (int)(fileSize - offset);
			val.line_offset = offset;
			val.line_size = blockSize;
			AddMapInputRecord(spec, &key, &val, sizeof(WC_KEY_T), sizeof(WC_VAL_T));	
			break;
		}
	}
	CUDA_SAFE_CALL(hipMemcpy(d_filebuf, h_filebuf, fileSize, hipMemcpyHostToDevice));	
	endTimer("preprocess", &preTimer);
	//----------------------------------------------
	//map/reduce
	//----------------------------------------------
	MapReduce(spec);

	endTimer("all", &allTimer);
	//----------------------------------------------
	//further processing
	//----------------------------------------------
#ifdef __OUTPUT__
	CUDA_SAFE_CALL(hipMemcpy(h_filebuf, d_filebuf, fileSize, hipMemcpyDeviceToHost));	
	validate(h_filebuf, spec, 10);
#endif
	//----------------------------------------------
	//finish
	//----------------------------------------------
	FinishMapReduce(spec);
	hipFree(d_filebuf);
	free(h_filebuf);

	return 0;
}
